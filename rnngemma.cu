#include "hip/hip_runtime.h"
# This is a port of the Pallad kernel from Recurrentgemma to CUDS

# https://github.com/google-deepmind/recurrentgemma/blob/main/recurrentgemma%2Fjax%2Fpallas.py

# There are some language differences between Pallas an CUDA , but a fairly straightforward mappinf of concepts 

# Pallas         CUDA
# kernel         kernel      - the code that runs on the specialzed hw
# task                        - the group of operations that can be done in parallel 
# data                        - the chunk of data to be worked on
# communication               - how the CPU and TPU share code and data
#                kernel       - the code that is excecuted on the GPU
#                __global__   - the keywod starting a kernel code block
#                threads
#                blocks
#                grids
#                memory 
#                 global       - cuda memory shared with cpu
#                 HMM          - on gPU memory shared among All SMs
#                 SM           - per processor memory  4k bytes or so, vert fast
#               __syncthreads__ - the checkpoint which synchronized per-thread execution globally 

# tasks are translated to kernels

# 
import functools
import math
from typing import NamedTuple

import torch
# import shard_map # i don't know what this is
import numpy as np
# array typing as at

class CudaShardingSoec(NamedTuple):
  """ the sharding spec for running a cuda kernel with shared values. """
  #mesh.jax.sharding